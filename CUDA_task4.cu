// Name: H.G. Manesha Washani
// Student Id: 1432289


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 20

__global__ void MatAdd(int A[][N], int B[][N], int C[][N]){
           int g = blockIdx.x;
           int h = blockIdx.y;

           C[g][h] = A[g][h] + B[g][h];
}

//int** randmatfunc();


void randmatfunc(int newmat[N][N]){
  int i, j, k; 
    for(i=0;i<N;i++){
        for(j=0;j<N;j++){
          k = rand() % 100 + 1;;
            printf("%d ", k);
            newmat[i][j] =k;
        }
        printf("\n");
       
    } 
  printf("\n--------------------------------------\n"); 
}

int main(){

int A[N][N];  
randmatfunc(A);
  
int B[N][N];  
randmatfunc(B);  



  int C[N][N];

  int (*d_A)[N], (*d_B)[N], (*d_C)[N];

  hipMalloc((void**)&d_A, (N*N)*sizeof(int));
  hipMalloc((void**)&d_B, (N*N)*sizeof(int));
  hipMalloc((void**)&d_C, (N*N)*sizeof(int));

  hipMemcpy(d_A, A, (N*N)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, (N*N)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_C, C, (N*N)*sizeof(int), hipMemcpyHostToDevice);

  int numThreads = 1;
  dim3 numBlocks(N,N);
  MatAdd<<<numBlocks,numThreads>>>(d_A,d_B,d_C);

  hipMemcpy(C, d_C, (N*N)*sizeof(int), hipMemcpyDeviceToHost);

  int g, h; printf("C = \n");
    for(g=0;g<N;g++){
        for(h=0;h<N;h++){
            printf("%d ", C[g][h]);
        }
        printf("\n");
    }

  hipFree(d_A); 
  hipFree(d_B); 
  hipFree(d_C);

  printf("\n");

  return 0;
}
