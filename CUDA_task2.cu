// Name: H.G. Manesha Washani
// Student Id: 1432289


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 4

/* The _global_ indicates a function that runs on the device and it called for host code. A kernel to add two integers */
 
__global__ void Matrix(int A[][N], int B[][N], int C[][N]){
           int g = threadIdx.x;
           int h = threadIdx.y;

           C[g][h] = A[g][h] + B[g][h];
}


int main(){

  int A[N][N] =
    {
      {1, 5, 6, 7},
      {4, 4, 8, 0},
      {2, 3, 4, 5},
      {2, 3, 4, 5}
   };

  int B[N][N] = 
    {
      {1, 5, 6, 7},
      {4, 4, 8, 0},
      {2, 3, 4, 5},
      {2, 3, 4, 5}
   };

  int C[N][N] = 
     {
      {0, 0, 0, 0},
      {0, 0, 0, 0},
      {0, 0, 0, 0},
      {0, 0, 0, 0}
   };

  
  int (*d_A)[N], (*d_B)[N], (*d_C)[N];

  /* device copies of A, B, C and Allocate space for device copies of A, B, C */
  
  hipMalloc((void**)&d_A, (N*N)*sizeof(int));
  hipMalloc((void**)&d_B, (N*N)*sizeof(int));
  hipMalloc((void**)&d_C, (N*N)*sizeof(int));
  
 // copy input to device

  hipMemcpy(d_A, A, (N*N)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, (N*N)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_C, C, (N*N)*sizeof(int), hipMemcpyHostToDevice);

  // Launch add() kernel on GPU
  int numBlocks = 1;
  dim3 threadsPerBlock(N,N);
  Matrix<<<numBlocks,threadsPerBlock>>>(d_A,d_B,d_C);

  // Copy result back to the host
  hipMemcpy(C, d_C, (N*N)*sizeof(int), hipMemcpyDeviceToHost);

  int g, h; printf("C = \n");
    for(g=0;g<N;g++){
        for(h=0;h<N;h++){
            printf("%d ", C[g][h]);
        }
        printf("\n");
    }

// This is cleanup 
  hipFree(d_A); 
  hipFree(d_B); 
  hipFree(d_C);

  printf("\n");

  return 0;
}

