// Name: H.G. Manesha Washani
// Student Id: 1432289


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 6

/* The _global_ indicates a function that runs on the device and it called for host code. A kernel to add two integers */

__global__ void MatAdd(int A[][N], int B[][N], int C[][N]){
           int g = threadIdx.x;
           int h = threadIdx.y;

           C[g][h] = A[g][h] + B[g][h];
}

/* My code i used variable is int g and int h, because of that reason i changed given code variables */


void randmatfunc(int newmat[N][N]){
  int g, h, k; 
    for(g=0;g<N;g++){
        for(h=0;h<N;h++){
          k = rand() % 100 + 1;;
            printf("%d ", k);
            newmat[g][h] =k;
        }
        printf("\n");
       
    } 
  printf("\n-----------------------------------\n"); 
}

int main(){

int A[N][N];  
randmatfunc(A);
  
int B[N][N];  
randmatfunc(B);  



  int C[N][N];

  int (*d_A)[N], (*d_B)[N], (*d_C)[N];

 /* device copies of A, B, C and Allocate space for device copies of A, B, C */
	
  hipMalloc((void**)&d_A, (N*N)*sizeof(int));
  hipMalloc((void**)&d_B, (N*N)*sizeof(int));
  hipMalloc((void**)&d_C, (N*N)*sizeof(int));

// copy input to device
 
  hipMemcpy(d_A, A, (N*N)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, (N*N)*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_C, C, (N*N)*sizeof(int), hipMemcpyHostToDevice);

// Launch add() kernel on GPU
  int numBlocks = 1;
  dim3 threadsPerBlock(N,N);
  MatAdd<<<numBlocks,threadsPerBlock>>>(d_A,d_B,d_C);

// Copy result back to the host
  hipMemcpy(C, d_C, (N*N)*sizeof(int), hipMemcpyDeviceToHost);

  int g, h; printf("C = \n");
    for(g=0;g<N;g++){
        for(h=0;h<N;h++){
            printf("%d ", C[g][h]);
        }
        printf("\n");
    }

// This is cleanup 
  hipFree(d_A); 
  hipFree(d_B); 
  hipFree(d_C);

  printf("\n");

  return 0;
}

